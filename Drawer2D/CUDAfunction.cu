#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "CUDAcomplex.cu"

#define PI 3.1415926535897932384626433832795f
#define twoPI 6.283185307179586476925286766559f
#define sqrt2PI 2.506628274631000502415765284811f

__global__ void CUDAJulia(float* xPos, float* yPos, uchar4* col, int iter, int xPixels, float4 f)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	//if (idx > num)
	//	return;
	hipComplex z(xPos[idx % xPixels], yPos[idx / xPixels]), c(f.x, f.y), c1(f.z, f.w);
	z = z.Inverse();
	int it = 0;
	while (it < iter && z.s() < 900.0f)
	{
		c = z*c1.exp()*c;
		//z = z*z + c1*z.Ln() + c*z.exp() +c*c1;
		//z = z*z*z.Ln()/c + c1/z;

		it++;
	}

	col[idx].z = char(127 * __cosf(it/266.0f-c1.x*10)) + 128;
	col[idx].y = char(127 * __cosf(it/15.0f-c1.y*10)) + 128;
	col[idx].x = char(127 * __cosf(it/4.0f-c1.x*10-c1.y*10)) + 128;
	col[idx].w = 122;
}

__host__ void StartCUDAJulia(float* xpos, float* ypos, uchar4* destcol, 
							 int xPixels, int yPixels, int iter, float4 f)
{
	float *xdev, *ydev;
	uchar4 *coldev;
	hipMalloc((void**)&xdev, xPixels * sizeof(float));
	hipMalloc((void**)&ydev, yPixels * sizeof(float));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));
	hipMemcpy((void*)xdev, (void*)xpos, xPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)ydev, (void*)ypos, yPixels * sizeof(float), hipMemcpyHostToDevice);

	CUDAJulia<<<dim3(xPixels * yPixels / 256), dim3(256)>>>(xdev, ydev, coldev, iter, xPixels, f);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		
	hipFree(xdev);
	hipFree(ydev);
	hipFree(coldev);
}


__global__ void CUDAJuliaMusic(float* xPos, float* yPos, uchar4* col, int iter, int xPixels, 
							   int* freq1, int* freq2, int anim)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	hipComplex z(xPos[idx % xPixels], yPos[idx / xPixels]), c(0, 0);
	int it = 0;
	//z.Inverse();
		while (it < iter && z.s() < 900.0f)
	{
		c.x = (freq1[it]);
		c.y = (freq2[it]);
		
		z = (z*z*z).exp() + hipComplex(-0.621f, 0.f);
		it++;
	}

	col[idx].z = 255-(char(127 * __cosf(it/266.0f)) + 128);
	col[idx].y = 255-(char(127 * __cosf(it/15.0f)) + 128);
	col[idx].x = 255-(char(127 * __cosf(it/4.0f)) + 128);
	col[idx].w = 122;
}

__host__ void StartCUDAJuliaMusic(float* xpos, float* ypos, uchar4* destcol, 
							 int xPixels, int yPixels, int iter,
							 int* freq1, int* freq2, int anim)
{
	float *xdev, *ydev;
	int *devfreq1, *devfreq2;
	uchar4 *coldev;

	hipMalloc((void**)&xdev, xPixels * sizeof(float));
	hipMalloc((void**)&ydev, yPixels * sizeof(float));
	hipMalloc((void**)&devfreq1, iter * sizeof(int));
	hipMalloc((void**)&devfreq2, iter * sizeof(int));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)xdev, (void*)xpos, xPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)ydev, (void*)ypos, yPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)devfreq1, (void*)freq1, iter * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*)devfreq2, (void*)freq2, iter * sizeof(int), hipMemcpyHostToDevice);

	CUDAJuliaMusic<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
		(xdev, ydev, coldev, iter, xPixels, devfreq1, devfreq2, anim);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		
	hipFree(xdev);
	hipFree(ydev);
	hipFree(coldev);
	hipFree(devfreq1);
	hipFree(devfreq2);
}


// ������ �������� ������
/*
z = z*z*z.Ln() + c;
z = z*z.Ln() + c;
z = z*z.exp() + z*z.Ln() + c;
z = z*z*z.Ln()/c + c;
z = z*z*z.Ln()/c + c/z; Invert colors
*/
//


__global__ void CUDAJuliaFFT(float* xPos, float* yPos, uchar4* col, int iter, int xPixels, 
							 hipComplex* c)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;


	hipComplex z(xPos[idx % xPixels], yPos[idx / xPixels]);
	int it = 0;
	//z = z.Inverse();
	while (it < iter && z.s() < 900.0f)
	{
		z = z*z + c[1];//c[0]*z.asin().exp() + c[1]*z.acos() + c[0]*z.asin(1) + c[1]*z.acos(1);
		it++;
	}

	col[idx].z = 255-(char(127 * __cosf(it/266.0f)) + 128);
	col[idx].y = 255-(char(127 * __cosf(it/15.0f)) + 128);
	col[idx].x = 255-(char(127 * __cosf(it/4.0f)) + 128);
	//col[idx].w = 122;
}

__host__ void StartCUDAJuliaFFT(float* xpos, float* ypos, uchar4* destcol, 
							 int xPixels, int yPixels, int iter,
							 int* freq)
{
	float *xdev, *ydev;
	float2 com[4];
	hipComplex *devCom;
	uchar4 *coldev;

	int S = 512/16;
	for(int j = 0; j < 4; j++)
	{
		com[j].x = 0;
		for(int i = 0; i < S; i++)
			com[j].x += freq[j*S + i + 8];
	}
	for(int j = 4; j < 8; j++)
	{
		com[j/2].y = 0;
		for(int i = 0; i < S; i++)
			com[j/2].y += freq[j*S + i + 8];
	}
	int max = 1, mid = 0;
	{
		if (com[0].x > max)
			max = com[0].x;
		if (com[1].x > max)
			max = com[1].x;
		if (com[2].x > max)
			max = com[2].x;
		if (com[3].x > max)
			max = com[3].x;
		if (com[0].y > max)
			max = com[0].y;
		if (com[1].y > max)
			max = com[1].y;
		if (com[2].y > max)
			max = com[2].y;
		if (com[3].y > max)
			max = com[3].y;
		
		mid += com[0].x;
		mid += com[1].x;
		mid += com[2].x;
		mid += com[3].x;
		mid += com[0].y;
		mid += com[1].y;
		mid += com[2].y;
		mid += com[3].y;
		mid >>= 3;
	}
	float x, y;
	max -= mid;
	for (int i = 0; i < 4; i++)
	{
		x = (com[i].x - mid) / max;
		y = (com[i].y - mid) / max;

		com[i].x = x;
		com[i].y = y;
	}

	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
	hipMalloc((void**)&xdev, xPixels * sizeof(float));
	hipMalloc((void**)&ydev, yPixels * sizeof(float));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)xdev, (void*)xpos, xPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)ydev, (void*)ypos, yPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);

	//hipMemcpy((void*)devfreq1, (void*)freq1, iter * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy((void*)devfreq2, (void*)freq2, iter * sizeof(int), hipMemcpyHostToDevice);

	CUDAJuliaFFT<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
		(xdev, ydev, coldev, iter, xPixels, devCom);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		
	hipFree(xdev);
	hipFree(ydev);
	hipFree(coldev);
	hipFree(devCom);
}

__device__ hipComplex func(hipComplex z, hipComplex* c)
{
	return z*z*z - 1.0f;
}

__device__ hipComplex der(hipComplex z, hipComplex* c, int k)
{
	//return z*z*3.0f;
	//return (func(zn, c) - func(zn_1, c))/(zn - zn_1);
	hipComplex dz(0.f,0.0001f);
	//dz.x = __cosf(k/50.0f) / 1000.0f;
	//dz.y = __sinf(k/50.0f) / 1000.0f;
	return (func(z+dz, c) - func(z, c))/dz;
}

/*
z = z*z*z.Ln() + c;
z = z*z.Ln() + c;
z = z*z.exp() + z*z.Ln() + c;
z = z*z*z.Ln()/c + c;
z = z*z*z.Ln()/c + c/z; Invert colors
*/

	__global__ void CUDANewton(float4 posData, uchar4* col, int iter, int xPixels, 
							 hipComplex* c, float k) // Nova
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hipComplex C(posData.y + posData.w*(idx / xPixels), posData.x + posData.w*(idx % xPixels)),
		z(c[0].x), zn;
	int it = 0;
	float s;
	do
	{
		zn = z;
		//z = z - ((-z*z).exp() - 1.0f)/C + c[1].x;
		//z = z*z*z.Ln() + C;
		//z = z*z.exp()*c[2].x + z*z.Ln()*c[1].x + C;
		//z = z - z(2.0f+c[1].x/10.f)*((z*z*z*z - 1.0f)/(z*z*z*4.0f - c[2].x)) + C;
		//z = z - z(2.0f)*((z*z*z - 1.0f)/(z*z*3.0f)) + C;
		//z = z - (z(0.7f + c[0].x*2.f)*(z*z*z - 1.0f)/(z*z*3.f)).pow(2) + C; 
		z = ((z*z + C) / (z*2.0f + C - 2)).pow(2) + 1 + c[1].x;
		//z = z - (z.exp()/z + C)/(z.exp()/z - z.exp()/(z*z));
		it++;
		s = (z-zn).s();
	}
	while (it < iter && s > 1.0e-5);// && s < 1.0e10);

	float color = it*2.0f + 10*c[0].x;
	col[idx].z = 255-(char(127 * __cosf(color/266.0f+3*c[1].x)) + 128);
	col[idx].y = 255-(char(127 * __cosf(color/15.0f+3*c[2].x)) + 128);
	col[idx].x = (char(127 * __cosf(color/4.0f+3*c[3].x)) + 128);
}

__global__ void CUDANewton_(float4 posData, uchar4* col, int iter, int xPixels, 
							 hipComplex* c, float k) // Magnetic
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hipComplex C(-posData.y - posData.w*(idx / xPixels), posData.x + posData.w*(idx % xPixels)),
		z(3*cosf(k/101)), zn;

	int it = 0;
	float s;
	do
	{
		zn = z;
		z = z - z(2.f)*((z*z*z - 1.f + 3*cosf(k/31))/(z*z*3.f + 3*cosf(k/61))) + C;
		it++;
		s = (z-zn).s();
	}
	while (it < iter && s > 0.00001f );

	float color = it*2.0f;
	col[idx].z = 255-(char(127 * __cosf(color/266.0f+3*c[1].x)) + 128);
	col[idx].y = (char(127 * __cosf(color/15.0f+3*c[2].x)) + 128);
	col[idx].x = (char(127 * __cosf(color/4.0f+3*c[3].x)) + 128);
}

__host__ void StartCUDANewton(float4 posData, uchar4* destcol, 
							 int xPixels, int yPixels, int iter,
							 int* freq)
{
	hipEvent_t start, startkernel;
	hipEvent_t stop, stopkernel;
	hipEventCreate(&start);
	hipEventCreate(&startkernel);
	hipEventCreate(&stop);
	hipEventCreate(&stopkernel);
	hipEventRecord(start, 0);

	float2 com[4];
	hipComplex *devCom;
	uchar4 *coldev;

	int S = 512/16;
	for(int j = 0; j < 4; j++)
	{
		com[j].x = 0;
		for(int i = 0; i < S; i++)
			com[j].x += freq[j*S + i + 6];
	}
	for(int j = 4; j < 8; j++)
	{
		com[j/2].y = 0;
		for(int i = 0; i < S; i++)
			com[j/2].y += freq[j*S + i + 6];
	}
	int max = 1, mid = 0;
	{
		if (com[0].x > max)
			max = com[0].x;
		if (com[1].x > max)
			max = com[1].x;
		if (com[2].x > max)
			max = com[2].x;
		if (com[3].x > max)
			max = com[3].x;
		if (com[0].y > max)
			max = com[0].y;
		if (com[1].y > max)
			max = com[1].y;
		if (com[2].y > max)
			max = com[2].y;
		if (com[3].y > max)
			max = com[3].y;
		
		mid += com[0].x;
		mid += com[1].x;
		mid += com[2].x;
		mid += com[3].x;
		mid += com[0].y;
		mid += com[1].y;
		mid += com[2].y;
		mid += com[3].y;
		mid >>= 3;
	}
	float x, y;
	//max -= mid;
	for (int i = 0; i < 4; i++)
	{
		x = (com[i].x - mid) / max;
		y = (com[i].y - mid) / max;

		com[i].x = x;
		com[i].y = y;
	}

	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);
	
	hipEventRecord(startkernel, 0);
	CUDANewton<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
		(posData, coldev, iter%256, xPixels, devCom, iter/256);
	hipEventRecord(stopkernel, 0);
	//hipEventSynchronize(stopkernel);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		
	hipFree(coldev);
	hipFree(devCom);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time, timekernel, without;
	hipEventElapsedTime(&time, start, stop);
	hipEventElapsedTime(&timekernel, startkernel, stopkernel);
	without = time - timekernel;
	hipEventDestroy(start);
	hipEventDestroy(startkernel);
	hipEventDestroy(stop);
	hipEventDestroy(stopkernel);
}

//__global__ void CUDANewton(float* xPos, float* yPos, uchar4* col, int iter, int xPixels, 
//							 hipComplex* c, float k)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	//hipComplex z(xPos[idx % xPixels], yPos[idx / xPixels]),zn(0.f,0.f);
//	hipComplex C(xPos[idx % xPixels], yPos[idx / xPixels]),z(1.f,0.f), zn;
//	//z = z*z*z - 1.0f;
//	//hipComplex /*f = func (z, c),*/ d = der(z, c, k);
//	//z = z.Inverse();
//	//z = z.Rotate(halfPI);
//	int it = 0;
//	float s;
//	do
//	{
//		//d = der(z, zn, c, k);
//		zn = z;
//		//z = z - (func (zn, c) / d)*2.0f;//er(z, c, k);
//		//z = func(z, c);
//		z = z - z(3.0f)*((z*z*z*z - 1.0f)/(z*z*z*4.0f)) + C;
//		//z = (z*z*z*2.0f + 1.0f)/(z*z*3.0f); 
//		//z = (z*z*z*2.0f)/(z*z*3.0f + c[0] * c[3]) + hipComplex(0.05f,c[0].y/10.0f).Rotate(k/PI);
//		//z = z*z + C;
//		//z = z - (z.ch() + c[it%4]);
//		it++;
//		//f = func (z, c);
//		//d = der (z, c, k);
//		s = (z-zn).s();
//	}
//	while (it < iter && s > 0.00001f && s < 10000.f);
//	
//	
//	//;// && z.s() < 16000.0f)
//
//	//idx*=4;
//	float color = it*2.0f;// + k/9.0f;
//	col[idx].z = (char(127 * __cosf(color/266.0f+3*c[1].x)) + 128);
//	col[idx].y = (char(127 * __cosf(color/15.0f+3*c[2].x)) + 128);
//	col[idx].x = (char(127 * __cosf(color/4.0f+3*c[3].x)) + 128);
//	//col[idx+3] = 122;
//}
//
//__host__ void StartCUDANewton(float* xpos, float* ypos, uchar4* destcol, 
//							 int xPixels, int yPixels, int iter,
//							 int* freq)
//{
//	hipEvent_t start, startkernel;
//	hipEvent_t stop, stopkernel;
//	hipEventCreate(&start);
//	hipEventCreate(&startkernel);
//	hipEventCreate(&stop);
//	hipEventCreate(&stopkernel);
//	hipEventRecord(start, 0);
//
//
//	float *xdev, *ydev;
//	float2 com[4];
//	hipComplex *devCom;
//	uchar4 *coldev;
//
//	int S = 512/16;
//	for(int j = 0; j < 4; j++)
//	{
//		com[j].x = 0;
//		for(int i = 0; i < S; i++)
//			com[j].x += freq[j*S + i + 8];
//	}
//	for(int j = 4; j < 8; j++)
//	{
//		com[j/2].y = 0;
//		for(int i = 0; i < S; i++)
//			com[j/2].y += freq[j*S + i + 8];
//	}
//	int max = 1, mid = 0;
//	{
//		if (com[0].x > max)
//			max = com[0].x;
//		if (com[1].x > max)
//			max = com[1].x;
//		if (com[2].x > max)
//			max = com[2].x;
//		if (com[3].x > max)
//			max = com[3].x;
//		if (com[0].y > max)
//			max = com[0].y;
//		if (com[1].y > max)
//			max = com[1].y;
//		if (com[2].y > max)
//			max = com[2].y;
//		if (com[3].y > max)
//			max = com[3].y;
//		
//		mid += com[0].x;
//		mid += com[1].x;
//		mid += com[2].x;
//		mid += com[3].x;
//		mid += com[0].y;
//		mid += com[1].y;
//		mid += com[2].y;
//		mid += com[3].y;
//		mid >>= 3;
//	}
//	float x, y;
//	//max -= mid;
//	for (int i = 0; i < 4; i++)
//	{
//		x = (com[i].x - mid) / max;
//		y = (com[i].y - mid) / max;
//
//		com[i].x = x;
//		com[i].y = y;
//	}
//
//	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
//	hipMalloc((void**)&xdev, xPixels * sizeof(float));
//	hipMalloc((void**)&ydev, yPixels * sizeof(float));
//	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));
//
//	hipMemcpy((void*)xdev, (void*)xpos, xPixels * sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy((void*)ydev, (void*)ypos, yPixels * sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);
//	
//	hipEventRecord(startkernel, 0);
//	CUDANewton<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
//		(xdev, ydev, coldev, iter%256, xPixels, devCom, iter/256);
//	hipEventRecord(stopkernel, 0);
//	//hipEventSynchronize(stopkernel);
//
//	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
//		
//	hipFree(xdev);
//	hipFree(ydev);
//	hipFree(coldev);
//	hipFree(devCom);
//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//
//	float time, timekernel, without;
//	hipEventElapsedTime(&time, start, stop);
//	hipEventElapsedTime(&timekernel, startkernel, stopkernel);
//	without = time - timekernel;
//	hipEventDestroy(start);
//	hipEventDestroy(startkernel);
//	hipEventDestroy(stop);
//	hipEventDestroy(stopkernel);
//}
/*
__global__ void CUDASeveral(float* xPos, float* yPos, uchar4* col, int iter, int xPixels, 
							 hipComplex* c, float k)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hipComplex z, zn; float s;

	z = hipComplex(xPos[idx % xPixels], yPos[idx / xPixels]);
	zn = hipComplex(0.f,0.f);
	int it1 = 0;
	do
	{
		zn = z;
		z = (z*z*z)*c[1] - 1.0f;
		it1++;
		s = (z-zn).s();
	}
	while (it1 < iter && s > 0.000001f && s < 1000000.f);

	z = hipComplex(xPos[idx % xPixels], yPos[idx / xPixels]);
	zn = hipComplex(0.f,0.f);
	int it2 = 0;
	do
	{
		zn = z;
		z = (z*z)*c[2]*c[0] - 1.0f;
		it2++;
		s = (z-zn).s();
	}
	while (it2 < iter && s > 0.000001f && s < 1000000.f);

	z = hipComplex(xPos[idx % xPixels], yPos[idx / xPixels]);
	zn = hipComplex(0.f,0.f);
	int it3 = 0;
	do
	{
		zn = z;
		z = (z*z*z*z)*c[3] - 1.0f;
		it3++;
		s = (z-zn).s();
	}
	while (it3 < iter && s > 0.000001f && s < 1000000.f);
	
	col[idx].z = 255-(char(127 * __cosf((it1+it2)/11.0f+c[0].y)) + 128);
	col[idx].y = 255-(char(127 * __cosf((it1+it3)/11.0f+c[1].y)) + 128);
	col[idx].x = 255-(char(127 * __cosf((it2+it3)/11.0f+c[2].y)) + 128);
}

__host__ void StartCUDASeveral(float* xpos, float* ypos, uchar4* destcol, 
							 int xPixels, int yPixels, int iter,
							 int* freq)
{
	hipEvent_t start, startkernel;
	hipEvent_t stop, stopkernel;
	hipEventCreate(&start);
	hipEventCreate(&startkernel);
	hipEventCreate(&stop);
	hipEventCreate(&stopkernel);
	hipEventRecord(start, 0);


	float *xdev, *ydev;
	float2 com[4];
	hipComplex *devCom;
	uchar4 *coldev;

	int S = 512/16;
	for(int j = 0; j < 4; j++)
	{
		com[j].x = 0;
		for(int i = 0; i < S; i++)
			com[j].x += freq[j*S + i + 8];
	}
	for(int j = 4; j < 8; j++)
	{
		com[j/2].y = 0;
		for(int i = 0; i < S; i++)
			com[j/2].y += freq[j*S + i + 8];
	}
	int max = 1, mid = 0;
	{
		if (com[0].x > max)
			max = com[0].x;
		if (com[1].x > max)
			max = com[1].x;
		if (com[2].x > max)
			max = com[2].x;
		if (com[3].x > max)
			max = com[3].x;
		if (com[0].y > max)
			max = com[0].y;
		if (com[1].y > max)
			max = com[1].y;
		if (com[2].y > max)
			max = com[2].y;
		if (com[3].y > max)
			max = com[3].y;
		
		mid += com[0].x;
		mid += com[1].x;
		mid += com[2].x;
		mid += com[3].x;
		mid += com[0].y;
		mid += com[1].y;
		mid += com[2].y;
		mid += com[3].y;
		mid >>= 3;
	}
	float x, y;
	max -= mid;
	for (int i = 0; i < 4; i++)
	{
		x = (com[i].x - mid) / max;
		y = (com[i].y - mid) / max;

		com[i].x = x;
		com[i].y = y;
	}

	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
	hipMalloc((void**)&xdev, xPixels * sizeof(float));
	hipMalloc((void**)&ydev, yPixels * sizeof(float));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)xdev, (void*)xpos, xPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)ydev, (void*)ypos, yPixels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);
	
	hipEventRecord(startkernel, 0);
	CUDASeveral<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
		(xdev, ydev, coldev, iter%256, xPixels, devCom, iter/256);
	hipEventRecord(stopkernel, 0);
	hipEventSynchronize(stopkernel);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
		
	hipFree(xdev);
	hipFree(ydev);
	hipFree(coldev);
	hipFree(devCom);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time, timekernel, without;
	hipEventElapsedTime(&time, start, stop);
	hipEventElapsedTime(&timekernel, startkernel, stopkernel);
	without = time - timekernel;
	hipEventDestroy(start);
	hipEventDestroy(startkernel);
	hipEventDestroy(stop);
	hipEventDestroy(stopkernel);
}
*/
struct data
{
	int iter;
	float k;
	int frac;
	unsigned int col;
};

//__device__ inline hipComplex Fracs(hipComplex z, hipComplex C, hipComplex*c, int frac, int k)
//{
//	switch (frac)
//	{
//	case 1:
//		return z - ((-z*z).exp() - 1.0f)/C + c[1].x;
//	case 2:
//		return z*z*z.Ln() + C;
//	case 3:
//		return z*z.exp()*c[2].x + z*z.Ln()*c[1].x + C;
//	case 4:
//		return z - z(2.0f+c[1].x/10.f)*((z*z*z*z - 1.0f)/(z*z*z*4.0f - c[2].x)) + C;
//	case 5:
//		return z - z(2.0f)*((z*z*z - 1.0f)/(z*z*3.0f)) + C;
//	case 6:
//		return z - (z(0.7f + c[0].x*2.f)*(z*z*z - 1.0f)/(z*z*3.f)).pow(2) + C; 
//	case 7:
//		return (z*z*z*2.0f)/(z*z*3.0f + c[0] * c[3]) + hipComplex(0.05f,c[0].y/10.0f).Rotate(k/PI);
//	case 8:
//		return z - (z.exp()/z + C)/(z.exp()/z - z.exp()/(z*z));
//	case 9:
//		return z.exp();
//	case 10:
//		return z.Ln();
//	case 11:
//		return z.sin();
//	case 12:
//		return z.ch();
//	case 13:
//		return z.asin();
//	case 14:
//		return z - z.exp() + C;
//	case 15:
//		return c[0] + c[1]*z + c[2]*z*z + c[3]*z*z*z + C;
//	case 16:
//		return z - z.sin() + C;
//	case 17:
//		return z - z.ch() + C;
//	case 18:
//		return z - z.sin()*(c[1].x+c[2].x) + C;
//	case 19:
//		return (-z*z).exp() + C;
//	case 20:
//		return (z+c[0].y)*c[0].x + (z/(C - c[2].y))*c[2].x + (z*z)*c[3].x + C;
//	case 21:
//		return hipComplex(__cosf(z.x), -__sinf(z.y));
//	case 22:
//		return c[0]*(z.i()*C*z).exp() + c[1]*(-z.i()*C*z).exp() + (C*C).Inverse(); 
//	case 23:
//		return (z+c[2].x).sin()*(z+c[1].x).cos() + C;
//	case 24:
//		return c[0]*((z*z).exp()) + c[1]*(z.exp()) + C + c[2]*z + c[3]*z*z;  
//	case 25:
//		return (-z*z).exp()*(1/(sqrt2PI)) + C;
//	default:
//		return ((z*z + C) / (z*2.0f + C - 2)).pow(2) + 1 + c[1].x;
//	}
//}

__device__ inline hipComplex Fracs(hipComplex z, hipComplex C, hipComplex*c, int frac, int k)
{
	switch (frac)
	{
	case 1:
		return z - ((-z*z).exp() - 1.0f)/C + c[1].x;
	case 2:
		return z*z*z.Ln() + C;
	case 3:
		return z*z.exp()*c[2].x + z*z.Ln()*c[3].x + C;
	case 4:
		return z - z(2.0f+c[0].y/10.f)*((z*z*z*z - 1.0f)/(z*z*z*4.0f - c[1].y)) + C;
	case 5:
		return z - z(2.0f)*((z*z*z - 1.0f)/(z*z*3.0f)) + C;
	case 6:
		return z - (z(0.7f + c[2].y*2.f)*(z*z*z - 1.0f)/(z*z*3.f)).pow(2) + C; 
	case 7:
		return (z*z*z*2.0f)/(z*z*3.0f + c[0] * c[3]) + hipComplex(0.05f,c[3].y/10.0f).Rotate(k/PI);
	case 8:
		return z - (z.exp()/z + C)/(z.exp()/z - z.exp()/(z*z));
	case 9:
		return (z+c[1].x).exp() + c[0];
	case 10:
		return (z+c[3].x).Ln() + c[2];
	case 11:
		return (z+c[2].x).sin() + c[1];
	case 12:
		return (z+c[0].x).ch() + c[3];
	case 13:
		return (z+c[2].x).acos() + c[1];
	case 14:
		return z - z.exp() + C;
	case 15:
		return c[0] + c[1]*z + c[2]*z*z + c[3]*z*z*z + C;
	case 16:
		return z - z.sin() + C;
	case 17:
		return z - z.ch() + C;
	case 18:
		return z - (z+c[1].d()).sin()*(c[0].y+c[2].x) + C + c[1].x;
	case 19:
		return (-z*z).exp() + C;
	case 20:
		return (z+c[0].y)*c[0].x + (z/(C - c[2].y))*c[2].x + (z*z)*c[3].x + C;
	case 21:
		return hipComplex(__cosf(z.x), -__sinf(z.y));
	case 22:
		return c[0]*(z.i()*C*z).exp() + c[1]*(-z.i()*C*z).exp() + (C*C).Inverse(); 
	case 23:
		return (z+c[2].x).sin()*(z+c[1].y).cos() + C;
	case 24:
		return c[0]*((z*z).exp()) + c[1]*(z.exp()) + C + c[2]*z + c[3]*z*z;  
	case 25:
		return (-z*z).exp()*(1/(sqrt2PI)) + C;
	case 26:
		return z - ((-z*z).exp() + c[2].x)/(-z(2)*z*(-z*z).exp()) + C;
	case 27:
		return z*z - ((-z*z).exp() + c[2].x)/(-z(2)*z*(-z*z).exp()) + C*C;
	case 28:
		return z*z + C;
	case 29:
		{
			hipComplex nz = z*z;
			float x = nz.x, y = nz.y;
			return z(3*x*x - 6*x*y - y*y, 3*x*x + 6*x*y - 3*y*y) + C;
		}
	case 30:
		return (c[0]-c[1]+c[2]-c[3])*z*(z(1)-z) + C;
	case 31:
		return z + z.cos()/z.sin() + C;
	case 32:
		{
			//hipComplex a1(c[0]), a2(-c[1]);
			//hipComplex	f1 = (z - a1)/(z(1) - z*a1.����������()),
			//			f2 = (z - a2)/(z(1) - z*a2.����������());
			//hipComplex f = (C*z.i()).exp()*f1*f2;
			return z - ((z-1)*(z-2)*(z-3))/((z+z.i())*(z+z.i()*2)*(z+z.i()*3)) + C;
		}
	default:
		return ((z*z + C + c[0].y) / (z*2.0f + C - 2 + c[2].y)).pow(2) + 1 + c[1].x;
	}
}

__device__ inline void function(hipComplex& z, hipComplex C, hipComplex* c, data st)
{
	z = Fracs(z, C, c, st.frac/1024, st.k);
	if (st.frac&1)
		z = Fracs(z, C, c, (st.frac%1024)/2, st.k);
}

__global__ void CUDAFractal(float4 posData, uchar4* pix, int xPixels, 
							hipComplex* c, data st, bool t)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
	float	x = posData.y + posData.w*(idy),
			y = posData.x + posData.w*(idx);
	hipComplex C, z, zn;
	if (t)
	{
		C = hipComplex(x, y);
		//float s = C.s();
		//z = C/s;
		z = hipComplex(c[0].x);
	}
	else
	{
		z = hipComplex(x, y);
		//float s = z.s();
		//C = z/s;
		C = hipComplex(c[0].x);
	}
	int it = 0;
	float s;
	do
	{
		zn = z;
		function(z, C, c, st);
		it++;
		s = (z-zn).s();
	}
	while (it < st.iter && s > 1.0e-6 && s < 1.0e9);
	float color = it*2.0f + 7*c[0].x + st.k;
	int r,g,b;
	//b = (char(127 * __cosf(color/166.0f+4*c[1].x)) + 128);
	//g = (char(127 * __cosf(color/15.0f+4*c[2].x)) + 128);
	//r = (char(127 * __cosf(color/4.0f+4*c[3].x)) + 128);
	b = (char(127 * __cosf(color/17.0f+4*c[1].x)) + 128);
	g = (char(127 * __cosf(color/11.0f+4*c[2].x)) + 128);
	r = (char(127 * __cosf(color/7.0f+4*c[3].x)) + 128);
	
	if (st.col&1) r = 255 - r;
	if (st.col&2) g = 255 - g;
	if (st.col&4) b = 255 - b;
	
	int pos = idx + idy*xPixels;
	pix[pos].x = r;
	pix[pos].y = g;
	pix[pos].z = b;/*
	pos = xPixels - idx + idy*xPixels;
	pix[pos].x = r;
	pix[pos].y = g;
	pix[pos].z = b;*/
}

__global__ void CUDAFractalOld(float4 posData, uchar4* pix, int xPixels, 
							hipComplex* c, data st, bool t)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;
	float	x = posData.y + posData.w*(idy),
			y = posData.x + posData.w*(idx);
	hipComplex C, z1, z2, z1n, z2n;
	
	if (t)
	{
		C = hipComplex(x, y);
		//float s = C.s();
		//z = C/s;
		z1 = hipComplex(c[0].x);
		z2 = hipComplex(c[1].x);
	}
	else
	{
		z1 = hipComplex(x, y);
		z2 = hipComplex(x, y);
		//float s = z.s();
		//C = z/s;
		C = hipComplex(c[0].x);
	}
	int it = 0;
	float s;
	do
	{
		z1n = z1;
		z2n = z2;
		z1 = z1 - (z2.pow(3) - 1)/(z1(3)*z1.pow(2)) + C;
		z2 = (z2 - z1.Inverse())/2 + C;
		
		it++;
		s = (z2-z2n).s();
	}
	while (it < st.iter && s > 1.0e-6);//&& s < 1.0e9);
	float color = it*2.0f + 7*c[0].x + st.k;
	int r,g,b;
	b = (char(127 * __cosf(color/166.0f+4*c[1].x)) + 128);
	g = (char(127 * __cosf(color/15.0f+4*c[2].x)) + 128);
	r = (char(127 * __cosf(color/4.0f+4*c[3].x)) + 128);
	
	if (st.col&1) r = 255 - r;
	if (st.col&2) g = 255 - g;
	if (st.col&4) b = 255 - b;
	
	int pos = idx + idy*xPixels;
	pix[pos].x = r;
	pix[pos].y = g;
	pix[pos].z = b;/*
	pos = xPixels - idx + idy*xPixels;
	pix[pos].x = r;
	pix[pos].y = g;
	pix[pos].z = b;*/
}

//__global__ void CUDAFractal(float4 posData, uchar4* pix, int xPixels, 
//							hipComplex* c, data st, bool t)
//{
//	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	float	x = posData.y + posData.w*(idx / xPixels),
//		y = posData.x + posData.w*(idx % xPixels);
//	hipComplex C, z, zn;
//	if (t)
//	{
//		C = hipComplex(x, y);
//		//z = C/c[0].x;
//		z = hipComplex(c[0].x);
//	}
//	else
//	{
//		z = hipComplex(x, y);
//		//C = z/c[0].x;
//		C = hipComplex(c[0].x);
//	}
//	int it = 0;
//	float s;
//	do
//	{
//		zn = z;
//		function(z, C, c, st);
//		it++;
//		s = (z-zn).s();
//	}
//	while (it < st.iter && s > 1.0e-5);// && s < 1.0e10);
//
//	float color = it*2.0f + 7*c[0].x + st.k;
//	int r,g,b;
//	b = (char(127 * __cosf(color/266.0f+4*c[1].x)) + 128);
//	g = (char(127 * __cosf(color/15.0f+4*c[2].x)) + 128);
//	r = (char(127 * __cosf(color/4.0f+4*c[3].x)) + 128);
//	
//	if (st.col&1) r = 255 - r;
//	if (st.col&2) g = 255 - g;
//	if (st.col&4) b = 255 - b;
//	
//	pix[idx].x = r;
//	pix[idx].y = g;
//	pix[idx].z = b;
//}
///<summary> 
/// ifct: Iter, Frac, Col, Type 
///</summary>
//__host__ void StartCUDAFractal(float4 posData, uchar4* destcol, 
//							   int xPixels, int yPixels, uint4 ifct, int* freq)
//{
//	float2 com[4];
//	hipComplex *devCom;
//	uchar4 *coldev;
//
//	int S = 40;
//	for(int j = 0; j < 4; j++)
//	{
//		com[j].x = 0;
//		for(int i = 0; i < S + 10; i++)
//			com[j].x += freq[j*S + i + 4];
//	}
//	for(int j = 4; j < 8; j++)
//	{
//		com[j/2].y = 0;
//		for(int i = 0; i < S + 10; i++)
//			com[j/2].y += freq[j*S + i + 4];
//	}
//	int max = 1, mid = 0;
//	{
//		if (com[0].x > max)
//			max = com[0].x;
//		if (com[1].x > max)
//			max = com[1].x;
//		if (com[2].x > max)
//			max = com[2].x;
//		if (com[3].x > max)
//			max = com[3].x;
//		if (com[0].y > max)
//			max = com[0].y;
//		if (com[1].y > max)
//			max = com[1].y;
//		if (com[2].y > max)
//			max = com[2].y;
//		if (com[3].y > max)
//			max = com[3].y;
//		
//		mid += com[0].x;
//		mid += com[1].x;
//		mid += com[2].x;
//		mid += com[3].x;
//		mid += com[0].y;
//		mid += com[1].y;
//		mid += com[2].y;
//		mid += com[3].y;
//		mid >>= 3;
//	}
//	float x, y;
//	//max -= mid;
//	if (max < 512)
//		com[0]=com[1]=com[2]=com[3]=float2();
//	else
//	for (int i = 0; i < 4; i++)
//	{
//		x = (com[i].x - mid) / max;
//		y = (com[i].y - mid) / max;
//
//		com[i].x = x;
//		com[i].y = y;
//	}
//
//	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
//	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));
//
//	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);
//
//	data st;
//	st.col = ifct.z;
//	st.frac = ifct.y;
//	st.iter = ifct.x%512;
//	st.k = ifct.x/512;
//	
//	CUDAFractal<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
//		(posData, coldev, xPixels, devCom, st, ifct.w);
//
//	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
//	
//	hipFree(devCom);
//	hipFree(coldev);
//}

__host__ void StartCUDAFractal(float4 posData, uchar4* destcol, 
							   int xPixels, int yPixels, uint4 ifct, int* freq)
{
	float2 com[4];
	hipComplex *devCom;
	uchar4 *coldev;

	int xx = 200;

	int S = 40;
	int i;
	com[0].x=com[1].x=com[2].x=com[3].x=
	com[0].y=com[1].y=com[2].y=com[3].y=0;
	for (i = 0; i < 7; i++)//8; i++)11
		if (freq[i]>xx)
		com[0].x += freq[i];//-xx;
	for (; i < 18+1; i++)//21; i++)29
		if (freq[i]>xx)
		com[1].x += freq[i];//-xx;
	i--;
	for (; i < 36+1; i++)//40; i++)58
		if (freq[i]>xx)
		com[2].x += freq[i];//-xx;
	i--;
	for (; i < 64+2; i++)//71; i++)103
		if (freq[i]>xx)
		com[3].x += freq[i];//-xx;
	i-=2;
	for (; i < 110+3; i++)//119; i++)178
		if (freq[i]>xx)
		com[0].y += freq[i];//-xx;
	i-=3;
	for (; i < 185+5; i++)//194; i++)299
		if (freq[i]>xx)
		com[1].y += freq[i];//-xx;
	i-=5;
	for (; i < 306+8; i++)//313; i++)495
		if (freq[i]>xx)
		com[2].y += freq[i];//-xx;
	i-=8;
	for (; i < 502+10; i++)//499; i++)812
		if (freq[i]>xx)
		com[3].y += freq[i];//-xx;
	int max = 1, mid = 0;
	{
		if (com[0].x > max)
			max = com[0].x;
		if (com[1].x > max)
			max = com[1].x;
		if (com[2].x > max)
			max = com[2].x;
		if (com[3].x > max)
			max = com[3].x;
		if (com[0].y > max)
			max = com[0].y;
		if (com[1].y > max)
			max = com[1].y;
		if (com[2].y > max)
			max = com[2].y;
		if (com[3].y > max)
			max = com[3].y;
		
		mid += com[0].x;
		mid += com[1].x;
		mid += com[2].x;
		mid += com[3].x;
		mid += com[0].y;
		mid += com[1].y;
		mid += com[2].y;
		mid += com[3].y;
		mid >>= 3;
	}
	float x, y;
	//max -= mid;
	max*=2;
	for (int i = 0; i < 4; i++)
	{
		x = (com[i].x - mid) / max;
		y = (com[i].y - mid) / max;

		com[i].x = x;
		com[i].y = y;
	}
	//float n = 10240;
	//if (max < n)
	//	for (int i = 0; i < 4; i++)
	//	{
	//		com[i].x *= (1-(n-max)/n);
	//		com[i].y *= (1-(n-max)/n);
	//	}
	//float n = (float)mid/(float)max;
	//if (n > 0.25)
	//	for (int i = 0; i < 4; i++)
	//	{
	//		com[i].x *= (1-n);
	//		com[i].y *= (1-n);
	//	}

	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);

	data st;
	st.col = ifct.z;
	st.frac = ifct.y;
	st.iter = ifct.x%512;
	st.k = ifct.x/512;
	
	//posData.x *= com[3].x;
	//posData.y *= com[3].x;
	//posData.w *= com[3].x;
	
	//CUDAFractal<<<dim3(xPixels*yPixels/256), dim3(256)>>>	
	CUDAFractal<<<dim3(xPixels/16, yPixels/16), dim3(16,16)>>>
		(posData, coldev, xPixels, devCom, st, ifct.w);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
	
	hipFree(devCom);
	hipFree(coldev);
}