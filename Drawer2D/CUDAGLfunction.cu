#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "CUDAcomplex.cu"

#define PI 3.1415926535897932384626433832795f
#define twoPI 6.283185307179586476925286766559f
#define sqrt2PI 2.506628274631000502415765284811f

struct data
{
	int iter;
	float k;
	int frac;
	unsigned int col;
};

__device__ inline hipComplex Fracs(hipComplex z, hipComplex C, hipComplex*c, int frac, int k)
{
	switch (frac)
	{
	case 1:
		return z - ((-z*z).exp() - 1.0f)/C + c[1].x;
	case 2:
		return z*z*z.Ln() + C;
	case 3:
		return z*z.exp()*c[2].x + z*z.Ln()*c[1].x + C;
	case 4:
		return z - z(2.0f+c[1].x/10.f)*((z*z*z*z - 1.0f)/(z*z*z*4.0f - c[2].x)) + C;
	case 5:
		return z - z(2.0f)*((z*z*z - 1.0f)/(z*z*3.0f)) + C;
	case 6:
		return z - (z(0.7f + c[0].x*2.f)*(z*z*z - 1.0f)/(z*z*3.f)).pow(2) + C; 
	case 7:
		return (z*z*z*2.0f)/(z*z*3.0f + c[0] * c[3]) + hipComplex(0.05f,c[0].y/10.0f).Rotate(k/PI);
	case 8:
		return z - (z.exp()/z + C)/(z.exp()/z - z.exp()/(z*z));
	case 9:
		return z.exp();
	case 10:
		return z.Ln();
	case 11:
		return z.sin();
	case 12:
		return z.ch();
	case 13:
		return z.asin();
	case 14:
		return z - z.exp() + C;
	case 15:
		return c[0] + c[1]*z + c[2]*z*z + c[3]*z*z*z + C;
	case 16:
		return z - z.sin() + C;
	case 17:
		return z - z.ch() + C;
	case 18:
		return z - z.sin()*(c[1].x+c[2].x) + C;
	case 19:
		return (-z*z).exp() + C;
	case 20:
		return (z+c[0].y)*c[0].x + (z/(C - c[2].y))*c[2].x + (z*z)*c[3].x + C;
	case 21:
		return hipComplex(__cosf(z.x), -__sinf(z.y));
	case 22:
		return c[0]*(z.i()*C*z).exp() + c[1]*(-z.i()*C*z).exp() + (C*C).Inverse(); 
	case 23:
		return (z+c[2].x).sin()*(z+c[1].x).cos() + C;
	default:
		return ((z*z + C) / (z*2.0f + C - 2)).pow(2) + 1 + c[1].x;
	}
}

__device__ inline void function(hipComplex& z, hipComplex C, hipComplex* c, data st)
{
	z = Fracs(z, C, c, st.frac/1024, st.k);
	if (st.frac&1)
		z = Fracs(z, C, c, (st.frac%1024)/2, st.k);
}

__global__ void CUDAFractal(float4 posData, uchar4* pix, int xPixels, 
							hipComplex* c, data st, bool t)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hipComplex C, z, zn;
	if (t)
	{
		C = hipComplex(posData.y + posData.w*(idx / xPixels), posData.x + posData.w*(idx % xPixels));
		//z = C/c[0].x;
		z = hipComplex(c[0].x);
	}
	else
	{
		z = hipComplex(posData.y + posData.w*(idx / xPixels), posData.x + posData.w*(idx % xPixels));
		//C = z/c[0].x;
		C = hipComplex(c[0].x);
	}
	int it = 0;
	float s;
	do
	{
		zn = z;
		function(z, C, c, st);
		it++;
		s = (z-zn).s();
	}
	while (it < st.iter && s > 1.0e-5);// && s < 1.0e10);

	float color = it*2.0f + 7*c[0].x + st.k;
	int r,g,b;
	b = (char(127 * __cosf(color/266.0f+4*c[1].x)) + 128);
	g = (char(127 * __cosf(color/15.0f+4*c[2].x)) + 128);
	r = (char(127 * __cosf(color/4.0f+4*c[3].x)) + 128);
	
	if (st.col&1) r = 255 - r;
	if (st.col&2) g = 255 - g;
	if (st.col&4) b = 255 - b;

	pix[idx].x = r;
	pix[idx].y = g;
	pix[idx].z = b;
}
///<summary> 
/// ifct: Iter, Frac, Col, Type 
///</summary>
__host__ void StartCUDAFractal(float4 posData, uchar4* destcol, 
							   int xPixels, int yPixels, uint4 ifct, int* freq)
{
	float2 com[4];
	hipComplex *devCom;
	uchar4 *coldev;

	int S = 40;
	for(int j = 0; j < 4; j++)
	{
		com[j].x = 0;
		for(int i = 0; i < S + 10; i++)
			com[j].x += freq[j*S + i + 4];
	}
	for(int j = 4; j < 8; j++)
	{
		com[j/2].y = 0;
		for(int i = 0; i < S + 10; i++)
			com[j/2].y += freq[j*S + i + 4];
	}
	int max = 1, mid = 0;
	{
		if (com[0].x > max)
			max = com[0].x;
		if (com[1].x > max)
			max = com[1].x;
		if (com[2].x > max)
			max = com[2].x;
		if (com[3].x > max)
			max = com[3].x;
		if (com[0].y > max)
			max = com[0].y;
		if (com[1].y > max)
			max = com[1].y;
		if (com[2].y > max)
			max = com[2].y;
		if (com[3].y > max)
			max = com[3].y;
		
		mid += com[0].x;
		mid += com[1].x;
		mid += com[2].x;
		mid += com[3].x;
		mid += com[0].y;
		mid += com[1].y;
		mid += com[2].y;
		mid += com[3].y;
		mid >>= 3;
	}
	float x, y;
	//max -= mid;
	if (max < 512)
		com[0]=com[1]=com[2]=com[3]=float2();
	else
	for (int i = 0; i < 4; i++)
	{
		x = (com[i].x - mid) / max;
		y = (com[i].y - mid) / max;

		com[i].x = x;
		com[i].y = y;
	}

	hipMalloc((void**)&devCom, 4 * sizeof(hipComplex));
	hipMalloc((void**)&coldev, xPixels * yPixels * sizeof(uchar4));

	hipMemcpy((void*)devCom, (void*)com, 4 * sizeof(float2), hipMemcpyHostToDevice);

	data st;
	st.col = ifct.z;
	st.frac = ifct.y;
	st.iter = ifct.x%512;
	st.k = ifct.x/512;
	
	CUDAFractal<<<dim3(xPixels * yPixels / 256), dim3(256)>>>
		(posData, coldev, xPixels, devCom, st, ifct.w);

	hipMemcpy((void*)destcol, (void*)coldev, xPixels * yPixels * sizeof(uchar4), hipMemcpyDeviceToHost);
	
	hipFree(devCom);
	hipFree(coldev);
}