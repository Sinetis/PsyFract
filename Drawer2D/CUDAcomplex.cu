#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#define PI 3.1415926535897932384626433832795f
#define twoPI 6.283185307179586476925286766559f
#define halfPI 1.5707963267948966192313216916398f

struct  hipComplex
{
	float x;
		float& Re() {return x;}
	float y;
		float& Im() {return y;}
		
	__device__ hipComplex(float re = 0, float im = 0)
		:x(re), y(im)
	{	}
	
	__device__ hipComplex(const hipComplex& val)
		:x(val.x), y(val.y)
	{	}

	__device__ inline hipComplex i()
	{return hipComplex(0.0f, 1.0f);}

	inline __device__ hipComplex operator= (hipComplex val)
	{
		this->x = val.x;
		this->y = val.y;
		return *this;
	}
	
	inline __device__ hipComplex operator+ (hipComplex val)
	{
		return hipComplex(this->x + val.x, this->y + val.y);
	}
	inline __device__ hipComplex operator+= (hipComplex val)
	{
		return *this = *this + val;
	}

	inline __device__ hipComplex operator- (hipComplex val)
	{
		return hipComplex(this->x - val.x, this->y - val.y);
	}
	inline __device__ hipComplex operator- ()
	{
		return hipComplex(-x, -y);
	}
	
	inline __device__ hipComplex operator* (hipComplex val)
	{
		// (x1 + iy1) * (x2 + iy2)
		hipComplex res;
		res.x = this->x * val.x - this->y * val.y;
		res.y = this->x * val.y + this->y * val.x;
		return res;
	}
	inline __device__ hipComplex operator*= (hipComplex val)
	{
		return *this = *this * val;
	}
	
	inline __device__ hipComplex operator/ (hipComplex val)
	{
		// (x1 + iy1) / (x2 + iy2)
		hipComplex res;
		float f = val.s();
		res.x = (this->x * val.x + this->y * val.y) / f;
		res.y = (-this->x * val.y + this->y * val.x) / f;
		return res;
	}

	inline __device__ hipComplex operator()(float re, float im = 0.f)
	{
		return hipComplex(re, im);
	}

	//inline __device__ operator float () const
	//{
	//	return sqrtf(this->x * this->x + this->y * this->y);
	//}
	// ������ �����
	inline __device__ float d(){return sqrtf(this->x * this->x + this->y * this->y);}
	// ������� ������ �����
	inline __device__ float s()
	{
		
		return this->x * this->x + this->y * this->y;}
	// �������� �����
	inline __device__ float arg()
	{
		if (x > 0)
			return atanf(y/x);
		if (y > 0)
			return atanf(y/x)+PI;
		else
			return atanf(y/x)-PI;
	}

	inline __device__ hipComplex operator +(float val)
	{
		return hipComplex(this->x+val, this->y);
	}

	inline __device__ hipComplex operator +=(float val)
	{
		this->x += val;
		return *this;
	}
	inline __device__ hipComplex operator -(float val)
	{
		return hipComplex(this->x-val, this->y);
	}

	inline __device__ hipComplex operator -=(float val)
	{
		this->x -= val;
		return *this;
	}

	inline __device__ hipComplex operator *(float val)
	{	
		return hipComplex(this->x*val, this->y*val);
	}

	inline __device__ hipComplex operator /(float val)
	{
		return hipComplex(this->x/val, this->y/val);
	}
	inline __device__ hipComplex operator /=(float val)
	{ return *this = *this / val; }

	inline __device__ hipComplex operator =(float val)
	{
		x = val;
		y = 0;
		return *this;
	}
	inline __device__ hipComplex operator^ (int val)
	{
		hipComplex c = *this;
		bool minus = val < 0;
		val = minus ? -val : val;
		for (int i = 1; i < val; i++) c *= *this;
		c = minus ? c.Inverse() : c;
		return c;
	}

	inline __device__ static hipComplex Zero()
	{
		return hipComplex(0,0);
	}

	// ������� � ��������
	inline __device__ hipComplex Rotate(float fi)
	{
		hipComplex res;
		float c = __cosf(fi), s = __sinf(fi);
		res.x = x*c - y*s;
		res.y = x*s + y*c;
		return res;
	}

	inline __device__ void Scale(float k)
	{
		this->x *= k;
		this->y *= k;
	}

	inline __device__ void Shift(hipComplex c)
	{
		Shift(c.x, c.y);
	}
	inline __device__ void Shift(float cx, float cy)
	{
		this->x += cx;
		this->y += cy;
	}

	inline __device__ hipComplex Inverse()
	{
		hipComplex c(x, y);
		float m = x*x + y*y;
		c.x /= m;
		c.y /= -m;
		return c;
	}

	inline __device__ hipComplex exp()
	{
		hipComplex c;
		float e;
		e = __expf(x); 
		c.x = e*__cosf(y);
		c.y = e*__sinf(y);
		return c;
	}

	inline __device__ hipComplex Ln()
	{
		hipComplex c;
		c.y = arg();
		c.x = __logf(this->d());
		return c;
	}
	
	inline __device__ hipComplex sin()
	{
		hipComplex c, eiz(0.f,1.f);
		eiz = (eiz * (*this)).exp(); 
		c = eiz - eiz.Inverse();
		c = c / hipComplex(0.f, 2.f);
		return c;
	}
	inline __device__ hipComplex cos()
	{
		hipComplex c, eiz(0.f,1.f);
		eiz = (eiz * (*this)).exp(); 
		c = eiz + eiz.Inverse();
		c = c / hipComplex(2.f, 0.f);
		return c;
	}

	inline __device__ hipComplex ch()
	{
		hipComplex c, ez(1.f,0.f);
		ez = (ez * (*this)).exp(); 
		c = ez + ez.Inverse();
		c = c / hipComplex(2.f, 0.f);
		return c;
	}
	inline __device__ hipComplex sh()
	{
		hipComplex c, ez(1.f,0.f);
		ez = (ez * (*this)).exp(); 
		c = ez - ez.Inverse();
		c = c / hipComplex(2.f, 0.f);
		return c;
	}

	inline __device__ hipComplex sqrt(bool root = false)
	{
		hipComplex c;
		float r = sqrtf(this->d());
		float a = (this->arg() + twoPI*root)/2;
		c.y = __sinf(a)*r;
		c.x = __cosf(a)*r;
		return c;
	}

	inline __device__ hipComplex asin(bool root = false)
	{
		hipComplex s = (-(*this)*(*this) + 1.f).sqrt(root);
		return hipComplex(0.f, -1.f)*(*this*hipComplex(0.f, 1.f) + s).Ln();
	}
	inline __device__ hipComplex acos(bool root = false, bool sign = true)
	{
		return hipComplex(halfPI, 0.f) - asin(root);
	}

	inline __device__ hipComplex atan()
	{
		hipComplex iz = (*this)*i();
		return ((-iz + 1.0f).Ln() - (iz + 1.0f).Ln())*(hipComplex(0.0f, 0.5f));
	}

	inline __device__ hipComplex pow(float a, float b = 0.0f)
	{
		float sr = x*x+y*y;
		float r, fi;
		fi = this->arg();
		r = powf(sr, a/2.0f)/(__expf(fi*b));
		fi = a*fi + b*__logf(sr)/2.0f;

		hipComplex res;
		res.x = r*__cosf(fi);
		res.y = r*__sinf(fi);
		return res;
	}
	inline __device__ hipComplex pow(hipComplex val)
	{
		return pow(val.x, val.y);
	}
	inline __device__ hipComplex pow(int val)
	{
		hipComplex c = *this;
		bool minus = val < 0;
		val = minus ? -val : val;
		for (int i = 1; i < val; i++) c *= *this;
		c = minus ? c.Inverse() : c;
		return c;
	}

	inline __device__ hipComplex ����������()
	{
		return hipComplex(x, -y);
	}
};